// 只是几个代码片段，通不过编译

// 应该是因为定义在了头文件里，所以用了static
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
 
 
#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

// check cuda error
inline void check(hipError_t call, const char* file, const int line)
{
    if (call != hipSuccess)
    {
        std::cout << "cuda error: " << hipGetErrorName(call) << std::endl;
        std::cout << "at file: " << file << ", line: " << line << std::endl;
        std::cout << hipGetErrorString(call) << std::endl;
    }
}
 
#define CHECK(call) (check(call, __FILE__, __LINE__))