
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime_api.h>

const int32_t n = 256;

__global__ void add_on_gpu(int32_t *a, int32_t *b, int32_t *c, int32_t n)
{
    // 0 <= threadIdx.x
    // warning #186-D: pointless comparison of unsigned integer with zero
    if(threadIdx.x < n) {
        c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
    }
}

int main()
{
    int32_t *a = new int32_t[n], *b = new int32_t[n], *c = new int32_t[n];

    for(int32_t i = 0; i < n; i += 1) {
        a[i] = i * i;
        b[i] = i * i * i;
    }

    int32_t *a_gpu, *b_gpu, *c_gpu;
    hipMalloc(&a_gpu, n * sizeof(int32_t));
    hipMalloc(&b_gpu, n * sizeof(int32_t));
    hipMalloc(&c_gpu, n * sizeof(int32_t));

    hipMemcpy(a_gpu, a, n * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, n * sizeof(int32_t), hipMemcpyHostToDevice);

    add_on_gpu<<<1,n>>>(a_gpu, b_gpu, c_gpu, n);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(1);
    }

    hipMemcpy(c, c_gpu, n * sizeof(int32_t), hipMemcpyDeviceToHost);

    bool found_error = false;
    for(int32_t i = 0; i < n; i += 1) {
        if(c[i] != a[i] + b[i]) {
            found_error = true;

            std::cout << "Error found: " << c[i] << " != " << a[i] << " + " << b[i] << std::endl;

            break;
        }
    }
    if(!found_error) std::cout << "Check passed" << std::endl;

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);

    delete[] a,b,c;
    return 0;
}